#include "hip/hip_runtime.h"
#include "colocation.cuh"
#include "spatial_distance.cuh"
#include "flow.hpp"
#include <vector>
#include <iostream>
#include "utils.hpp"
#include "feature_set.hpp"
#include <algorithm>

using namespace std;

#define SUM_BLK_SIZE 128

/*
Take intersection of two sets of neighbors
Excludes neighbors who are in the excluded class list
*/
void merge_neighbors(int * combined_list, int * list_1, int * list_2, int * class_lookup, int max_degree,int class_filter)
{
    int index = 0;
    for(int i=0;i<max_degree;i++)
    {
        int candidate = list_1[i];
        //End of list
        if(candidate==-1){break;}
        if(class_lookup[candidate]!=class_filter)
        {
            bool mutual = false;
            for(int j=0;j<max_degree;j++)
            {
                int query = list_2[j];
                if(query==-1){break;}
                if(query==candidate){mutual=true;break;}
            }
            if(mutual)
            {
                combined_list[index] = candidate;
                index++;
            }
        }
    }
}

int build_fclp(int * members,int * features, int * neighbors,int * members_,int * features_,int * neighbors_,int * adj_list, int k, int pair_count, int max_degree,int * class_lookup)
{
    int index = 0;
    for(int i=0;i<pair_count;i++)
    {
        for(int j=0;j<max_degree;j++)
        {
            int neighbor = neighbors[i*max_degree + j];
            //No neighbor case
            if(neighbor==-1)
            {
                break;
            }
            
            for(int w=0;w<k-1;w++)
            {
                members_[index*k+w] = members[i*(k-1)+w];
                features_[index*k+w] = features[i*(k-1)+w];
            }
            members_[index*k+k-1] = neighbor;
            features_[index*k+k-1] = class_lookup[neighbor];
            merge_neighbors(neighbors_+index*max_degree,neighbors+i*max_degree,adj_list+neighbor*max_degree,class_lookup,max_degree,class_lookup[neighbor]);
            index++;      
        }
    }
    return index;
}

/*
Function to remove patterns with a FCI below specified threshold
*/
int purge_fclp(int *members,int *features,int *neighbors,float frequency_threshold,int k,int max_degree,int table_length, int * class_frequency)
{
    vector<FeatureSet> feature_patterns;
    vector<int> feature_patterns_count;
    int feature_index[table_length];
    vector<int> bad_features;

    for(int i=0;i<table_length;i++)
    {
        FeatureSet set = FeatureSet(features+i*k,k);
        bool in_set = false;
        for(int j=0;j<feature_patterns.size();j++){
            if (feature_patterns[j].is_equivalent(set))
            {
                in_set = true;
                feature_patterns_count[j] = feature_patterns_count[j] + 1;
                feature_index[i] = j;
            }
        }
        if(!in_set)
        {
            feature_patterns.push_back(set);
            feature_patterns_count.push_back(0);
            feature_index[i] = feature_patterns.size()-1;
        }
    }

    for(int i=0;i<feature_patterns.size();i++)
    {
        FeatureSet set = feature_patterns[i];
        float fci = (float)feature_patterns_count[i]/class_frequency[set.features[0]];
        for(int j=1;j<k;j++)
        {
            fci = min((float)feature_patterns_count[i]/class_frequency[set.features[j]],fci);
        }
        cout << fci << endl;
        if(fci<frequency_threshold)
        {
            bad_features.push_back(i);
        }
    }

    int idx = 0;
    for(int i=0;i<table_length;i++)
    {
        if (find(bad_features.begin(), bad_features.end(), feature_index[i]) != bad_features.end()) {
            //do nothing
        }
        else {
            //copy row to first open spot and increment table length
            for(int w=0;w<k;w++)
            {
                members[idx*k+w] = members[i*k+w];
                features[idx*k+w] = features[i*k+w];
            }
            for(int w=0;w<max_degree;w++)
            {
                neighbors[idx*max_degree+w] = neighbors[i*max_degree+w];
            }
            idx++;
        }
    }
    return idx;
}


/*
Function to get the size of the k+1 table
*/
int get_max_pair_count(int * neighbors,int length,int max_degree)
{
    int pair_count = 0;
    for(int i=0;i<length;i++)
    {
        for(int j=0;j<max_degree;j++)
        {
            int neighbor = neighbors[max_degree*i + j];
            if(neighbor>-1)
            {
                pair_count++;
            }
        }
    }
    return pair_count;
}

/*
Given a threshold value converts every distance in matrix 
to
 0 - not neighbors
 1 - neighbors
Essentially creates incidence matrix
*/
__global__ void threshold_distance_matrix(float * dist_matrix,float threshold,int length){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i>=length*length)
    {
        return;
    }
    if(dist_matrix[i]>threshold)
    {
        dist_matrix[i] = 0;
    }
    else
    {
        dist_matrix[i] = 1;
    }
}

__global__ void get_neighbors(float * dist_matrix,int * neighbor_table, int max_degree ,int length){
	
    // Index number
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i>=length)
    {
        return;
    }
    int row_index = i * length;
    int idx = 0;
    for(int j=0;j<length;j++)
    {
        if(dist_matrix[row_index + j]>0 && j!=i)
        {
            neighbor_table[i*max_degree + idx] = j;
            idx++;
        }
    }
    __syncthreads();
}

__global__ void sum_rows(float * matrix,int * sum_arr,int length)
{
    __shared__ float partial_sum[2*SUM_BLK_SIZE];

    int row = (int)((blockIdx.x*blockDim.x)/length);

    unsigned int t = threadIdx.x;
    unsigned int start = 2*blockIdx.x*blockDim.x;

    if(blockIdx.x*blockDim.x+t >=length*length)
    {
        return;
    }
    if(start + t >=(row+1)*length)
    {
        partial_sum[t] = 0;
    }
    else
    {
        partial_sum[t] = matrix[start + t];
    }

    if(blockDim.x+t >=(row+1)*length)
    {
        partial_sum[blockDim.x+t] = 0;
    }
    else
    {
        partial_sum[blockDim.x+t] = matrix[start + blockDim.x+t];
    }
    for (unsigned int stride = blockDim.x; 
        stride > 0;  stride /= 2) 
    {
        __syncthreads();
        if (t < stride)
        partial_sum[t] += partial_sum[t+stride];
    }
    __syncthreads();

    if(t==0)
    {
        atomicAdd(sum_arr+row,(int)partial_sum[0]);
    }
    __syncthreads();

}

ColocationResult colocate(vector<FlowData> flows,float frequency_threshold, float spatial_threshold, size_t shared_mem_size)
{

    //Ensure we have a fresh device
    hipDeviceReset();
    
    //Determine total number of points
    int length = 0;
    for(int i=0;i<flows.size();i++)
    {
        length += flows[i].length;
    }

    // copy data to GPU 
	float *d_sx, *d_sy, *d_dx, *d_dy, *d_L;
	hipMalloc((void**)&d_sx, length*sizeof(float));
	hipMalloc((void**)&d_sy, length*sizeof(float));
	hipMalloc((void**)&d_dx, length*sizeof(float));
	hipMalloc((void**)&d_dy, length*sizeof(float));
	hipMalloc((void**)&d_L, length*sizeof(float));
    int * class_lookup = (int *)malloc(length*sizeof(int));
    int * index_lookup = (int *)malloc(length*sizeof(int));
    int * class_frequency = (int * )malloc(flows.size()*sizeof(int));
    for(int i=0;i<flows.size();i++){class_lookup[i] = 0;}

    int start_idx = 0;
    for(int i=0;i<flows.size();i++)
    { 
        FlowData flow = flows[i];
        hipMemcpy(d_sx + start_idx, flow.sx, flow.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_sy + start_idx, flow.sy, flow.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_dx + start_idx, flow.dx, flow.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_dy + start_idx, flow.dy, flow.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_L + start_idx, flow.L, flow.length*sizeof(float), hipMemcpyHostToDevice);
        // printf("GPUassert: %s \n", hipGetErrorString(hipGetLastError()));

        for(int j=0;j<flow.length;j++)
        {
            class_lookup[start_idx+j] = i;
            index_lookup[start_idx+j] = j;
            class_frequency[i]++;
        }
        start_idx += flow.length;
    }

    float * dist_matrix_gpu;
	hipMalloc((void**)&dist_matrix_gpu, length*length*sizeof(float));
	hipMemset(dist_matrix_gpu, 0, length*length*sizeof(float));

    calculate_spatial_distance_matrix <<< ceil((float)length/128) , 128, shared_mem_size >>> (d_sx,d_sy,d_dx,d_dy,d_L,dist_matrix_gpu,length,2,1.0f);
    threshold_distance_matrix <<< ceil((float)length*length/128) , 128 >>>(dist_matrix_gpu,spatial_threshold,length);
    hipFree((void**)&d_sx);
    hipFree((void**)&d_sy);
    hipFree((void**)&d_dx);
    hipFree((void**)&d_sx);
    hipFree((void**)&d_dy);
    
	
    // Get maximum degree of flow neighbor graph
    int * number_neighbors;
    hipMalloc((void**)&number_neighbors, length*sizeof(int));
	hipMemset(number_neighbors, 0, length*sizeof(int));

    // sum_rows <<< ceil((float)(length*length)/(2*128)) , 128 >>>(dist_matrix_gpu,number_neighbors,length);

    float * dist_matrix_cpu = (float*)malloc(length*length*sizeof(float));
	hipMemcpy(dist_matrix_cpu, dist_matrix_gpu, length*length*sizeof(float), hipMemcpyDeviceToHost);

    int max_degree = 0;
    long int total = 0;
    for(int i=0;i<length;i++)
    {
        int tmp = 0;
        for(int j=0;j<length;j++)
            tmp += dist_matrix_cpu[i*length + j];
            total++;
        if(tmp>max_degree)
            max_degree = tmp;
    }
    
    /*
    Initialize adjacency list structure
    */
    int *adj_list_cpu;
    // hipMalloc((void**)&adj_list_gpu, length*max_degree*sizeof(int));
	// hipMemset(number_neighbors, -1, length*max_degree*sizeof(int));
    adj_list_cpu = (int*)malloc(length*max_degree*sizeof(int));
    // get_neighbors<<< ceil((float)length/128) , 128 >>>(dist_matrix_gpu,adj_list_gpu, max_degree ,length);
    for(int i=0;i<length*max_degree;i++)
        {adj_list_cpu[i] = -1;}
    for(int i=0;i<length;i++)
    {
        int row = i*length;
        int index = 0;
        for(int j=0;j<max_degree;j++)
        {
            if(dist_matrix_cpu[row+j]>0)
            {
                if(class_lookup[j]!=class_lookup[i])
                {
                    adj_list_cpu[i*max_degree + index] = j;
                    index++;
                }
            }
        }
    }

    // hipFree(dist_matrix_gpu);
	// hipMemcpy(adj_list_cpu, adj_list_gpu, length*max_degree*sizeof(int), hipMemcpyDeviceToHost);
    
    /* Initialize k=1 table
    */
    int * members_, * features_, *neighbors_;
    int *members, *features, *neighbors;

    members = (int *)malloc(length*sizeof(int));
    features = (int *)malloc(length*sizeof(int));
    neighbors = (int *)malloc(max_degree*length*sizeof(int));

    for(int i=0;i<max_degree*length;i++){neighbors[i] = -1;}
    int pair_count = 0;
    
    for(int i=0;i<length;i++)
    {
        int row = i*max_degree;
        int index = 0;
        members[i] = i;
        features[i] = class_lookup[i];
        for(int j=0;j<max_degree;j++)
        {
            if(adj_list_cpu[row + j]>i)
            {
                pair_count++;
                neighbors[i*max_degree+index] = adj_list_cpu[row+j];
                index++;
            }
        }
    }


    /*
    Iteratively build FCLP tables
    */
    int pair_count_old=length;
    int k_val=1;
    for(int k=2;k<flows.size()+1;k++)
    {
        cout << k << " " << flows.size() << endl;
        if(k>2)
        {
            free(members_);
            free(features_);
            free(neighbors_);
        }
        members_ = members;
        features_ = features;
        neighbors_ = neighbors;
        members = (int *)malloc(k*pair_count*sizeof(int));
        features = (int *)malloc(k*pair_count*sizeof(int));
        neighbors = (int *)malloc(max_degree*pair_count*sizeof(int));
        cout << max_degree*pair_count*sizeof(int) << endl;
        for(int i=0;i<max_degree*pair_count;i++){neighbors[i]=-1;}
        int table_length = build_fclp(members_,features_,neighbors_,members,features,neighbors,adj_list_cpu,k,pair_count_old,max_degree,class_lookup);
        table_length = purge_fclp(members,features,neighbors,frequency_threshold,k,max_degree,table_length,class_frequency);
        pair_count_old=pair_count;
        pair_count = get_max_pair_count(neighbors,table_length,max_degree);
        if(pair_count==0)
        {
            if(table_length>0)
            {
                pair_count_old = table_length;
                free(members_);
                free(features_);
                free(neighbors_);
                members_ = members;
                features_ = features;
                neighbors_ = neighbors;
            }
            break;
        }else{k_val++;}
    }
    ColocationResult result(k_val);
    result.indices = members_;
    result.class_lookup = class_lookup;
    result.index_lookup = index_lookup;
    result.length = k_val*pair_count_old;
    result.flow_length = length;
    return result;
}