#include "hip/hip_runtime.h"
/*
Cole Hill
University of South Florida
Computer Science and Engineering
Programming Massively Parallel Systems
Fall 2021
*/
#include <iostream>
#include <math.h>
#include "flow_k.cuh"
#include "spatial_distance.cuh"
#include "flow.hpp"
#include <algorithm>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include "utils.hpp"

#define PI 3.1416
#define BLOCK_DIM 128


/*
Implements ipleys K-function adapted to flows
Calculation taken from http://ceadserv1.nku.edu/longa//geomed/ppa/doc/localK/localK.htm

distance_matrix is a precomputed distance matrix between all flows
*/
__device__ float local_k_function(float * distance_matrix,int index, float r,float A,int length)
{   
    //Count number of flows located within spatial radius threshold r
    int count = 0;
    for(int i=0;i<length;i++)
    {
        if(distance_matrix[index*length + i]<r)
        {
            count++;
        }
    }
    count--; // ignore self
    //return k value
    return sqrt(A*count/(PI*length-1));
}

/*
Implements cross K-function adapted to flows
Flow Cross K-function: a bivariate flow analytical method
https://onlinelibrary.wiley.com/doi/epdf/10.1111/gean.12100

distance_matrix is a precomputed distance matrix between all flows
piv is index where classes switch
*/
__device__ float local_cross_k_function(float * distance_matrix,int index, float r,float A,int length,int piv)
{
    int count = 0;
    int start,end;
    //check first half of matrix
    if(index>piv)
    {
        start = 0;
        end = piv;
    }
    //check second half of matrix
    else
    {
        start = piv;
        end = length;
    }

    //Calculate ripley's k function
    for(int i=start;i<end;i++)
    {
        if(distance_matrix[index*length + i]<r)
        {
            count++;
        }
    }
    return sqrt(A*count/(PI*length-1));
}

/*
GPU kernel for calculating K funtion for all flows
*/
__global__ void calculate_k_function(float * dist_matrix, float * sig_vec, float r,float A, int length){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i>=length)
        return;
    float sig_value = local_k_function(dist_matrix,i,r,A,length);
    sig_vec[i] = sig_value;
}

/*
GPU kernel for calculating cross-K funtion for all flows
*/
__global__ void calculate_cross_k_function(float * dist_matrix, float * sig_vec, float r,float A, int length, int piv){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i>=length)
        return;
    float sig_value = local_cross_k_function(dist_matrix,i,r,A,length,piv);
    sig_vec[i] = sig_value;
}

/*
GPU kernel for shuffling vectors
*/
__global__ void vec_shuffle(float * vec, int length){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i>=length){
        return;
    }
    // Init random number generator
    hiprandState state;
    hiprand_init((unsigned long long)clock() + i, 0, 0, &state);
    
    // Pick alternate value to take
    int j = (int)(hiprand_uniform(&state)*(length-1));
    float eps = hiprand_uniform(&state)*0.1 - 0.05;
    float value = vec[j] + eps;
    __syncthreads();
    //Update value
    vec[i] = value;
    __syncthreads();
}

/*
GPU kernel for shuffling class labels of flows
*/
__global__ void shuffle_label(float * d_sx, float * d_sy, float * d_dx, float * d_dy, float * d_L, int length){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i>=length){
        return;
    }
    // Init random number generator
    hiprandState state;
    hiprand_init((unsigned long long)clock() + i, 0, 0, &state);
    
    // Pick alternate value to take
    int j = (int)(hiprand_uniform(&state)*(length-1));
    float eps = hiprand_uniform(&state)*0.1 - 0.05;
    float sx,sy,dx,dy,L;
    sx = d_sx[j];
    sy = d_sy[j];
    dx = d_dx[j];
    dy = d_dy[j];
    L = d_L[j];
    //Update value
    __syncthreads();
    d_sx[i] = sx;
    d_sy[i] = sy;
    d_dx[i] = dx;
    d_dy[i] = dy;
    d_L[i] = L;
    __syncthreads();
}

/*
Calculate Ripleys k value for each flow and run monte carlo simulation to 
test significance. Returns significant flows
*/
std::vector<int> process_flow_k(FlowData flow, float *d_sx, float *d_sy, float *d_dx, float *d_dy, float *d_L, float * dist_matrix_gpu, int num_iter, int func_type,float alpha,size_t shared_mem_size,float radius)
{ 
    // K-value vectors
    float * sig_vec_real;
    float * sig_vec_cpu;
    sig_vec_real = (float *)malloc(sizeof(float)*flow.length);
    sig_vec_cpu = (float *)malloc(sizeof(float)*flow.length);

    // Copy flow data to GPU
    hipMemcpy(d_sx, flow.sx, flow.length*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sy, flow.sy, flow.length*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dx, flow.dx, flow.length*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dy, flow.dy, flow.length*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_L, flow.L, flow.length*sizeof(float), hipMemcpyHostToDevice);
    // Calculate spatial distance matrix
    calculate_spatial_distance_matrix <<< ceil((float)flow.length/BLOCK_DIM) , BLOCK_DIM, shared_mem_size >>> (d_sx,d_sy,d_dx,d_dy,d_L,dist_matrix_gpu,flow.length,func_type,alpha);

    // Calculate k values for real data
    float * sig_vec;
    hipMalloc((void**)&sig_vec, flow.length*sizeof(float));
    calculate_k_function<<< ceil((float)flow.length/128) , 128 >>>(dist_matrix_gpu,sig_vec,radius,flow.area,flow.length);
	hipMemcpy(sig_vec_real, sig_vec, flow.length*sizeof(float), hipMemcpyDeviceToHost);

    //Generate synthetic data (monte carlo step)
    float upper_envelope, lower_envelope;
    for (int ii=0;ii<num_iter;ii++)
    {
        // Copy flow data to GPU
        hipMemcpy(d_sx, flow.sx, flow.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_sy, flow.sy, flow.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_dx, flow.dx, flow.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_dy, flow.dy, flow.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_L, flow.L, flow.length*sizeof(float), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        // Shuffle endpoints
        vec_shuffle<<< ceil((float)flow.length/64) , 64 >>>(d_sx, flow.length);
        hipDeviceSynchronize();
        vec_shuffle<<< ceil((float)flow.length/64) , 64 >>>(d_sy, flow.length);
        vec_shuffle<<< ceil((float)flow.length/64) , 64 >>>(d_dx, flow.length);
        vec_shuffle<<< ceil((float)flow.length/64) , 64 >>>(d_dy, flow.length);
        // Calculate spatial distance matrix
        calculate_spatial_distance_matrix <<< ceil((float)flow.length/128) , 128, shared_mem_size >>> (d_sx,d_sy,d_dx,d_dy,d_L,dist_matrix_gpu,flow.length,alpha,func_type);
        // Calculate k values for synthetic data
        calculate_k_function<<< ceil((float)flow.length/128) , 128 >>>(dist_matrix_gpu,sig_vec,radius,flow.area,flow.length);
        hipMemcpy(sig_vec_cpu, sig_vec, flow.length*sizeof(float), hipMemcpyDeviceToHost);
        // Find upper and lower significance values
        if(ii>0)
        {
            upper_envelope = max(upper_envelope,get_max(sig_vec_cpu,flow.length));
            lower_envelope = min(lower_envelope,get_min(sig_vec_cpu,flow.length));
        }
        else
        {
            upper_envelope = get_max(sig_vec_cpu,flow.length);
            lower_envelope = get_min(sig_vec_cpu,flow.length);
        }
    }
    //Return flows with k-value above significance threshold
    std::vector<int> output;
    for(int i=0;i<flow.length;i++)
    {
        if(sig_vec_real[i]>upper_envelope)
        {
            output.push_back(i);
        }
    }
    return output;
}

/*
Calculate cross flow Ripleys k value for each flow and run monte carlo simulation to 
test significance. Returns significant flows
*/
std::vector<int> process_cross_flow_k(FlowData flow_1, FlowData flow_2, int num_iter, int func_type,float alpha,size_t shared_mem_size,float radius)
{ 
    
    hipDeviceReset();
    
    float * sig_vec_real;
    float * sig_vec_cpu;
    int length = flow_1.length + flow_2.length;

    // copy data to GPU 
	float *d_sx, *d_sy, *d_dx, *d_dy, *d_L;
	hipMalloc((void**)&d_sx, length*sizeof(float));
	hipMalloc((void**)&d_sy, length*sizeof(float));
	hipMalloc((void**)&d_dx, length*sizeof(float));
	hipMalloc((void**)&d_dy, length*sizeof(float));
	hipMalloc((void**)&d_L, length*sizeof(float));
    //Copy first flow
	hipMemcpy(d_sx, flow_1.sx, flow_1.length*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_sy, flow_1.sy, flow_1.length*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dx, flow_1.dx, flow_1.length*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dy, flow_1.dy, flow_1.length*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_L, flow_1.L, flow_1.length*sizeof(float), hipMemcpyHostToDevice);
    //Copy second flow
    hipMemcpy(d_sx + flow_1.length, flow_2.sx, flow_2.length*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_sy + flow_1.length, flow_2.sy, flow_2.length*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dx + flow_1.length, flow_2.dx, flow_2.length*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dy + flow_1.length, flow_2.dy, flow_2.length*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_L + flow_1.length, flow_2.L, flow_2.length*sizeof(float), hipMemcpyHostToDevice);
    
    // K-value vectors
    sig_vec_real = (float *)malloc(sizeof(float)*length);
    sig_vec_cpu = (float *)malloc(sizeof(float)*length);
    
    // Calculate study area
    float max_x = max(flow_1.max_x,flow_2.max_x);
    float max_y = max(flow_1.max_y,flow_2.max_y);
    float min_x = min(flow_1.min_x,flow_2.min_x);
    float min_y = min(flow_1.min_y,flow_2.min_y);
    float A = (max_x-min_x)*(max_y-min_y);

    // Calculate spatial distance matrix
    float * dist_matrix_gpu;
	hipMalloc((void**)&dist_matrix_gpu, length*length*sizeof(float));
	hipMemset(dist_matrix_gpu, 0, length*length*sizeof(float));
    calculate_spatial_distance_matrix <<< ceil((float)length/128) , 128, shared_mem_size >>> (d_sx,d_sy,d_dx,d_dy,d_L,dist_matrix_gpu,length,func_type,alpha);

    // Calculate k values for real data
    float * sig_vec;
    hipMalloc((void**)&sig_vec, length*sizeof(float));
    calculate_cross_k_function<<< ceil((float)length/128) , 128 >>>(dist_matrix_gpu,sig_vec,radius,A,length,flow_1.length);
	hipMemcpy(sig_vec_real, sig_vec, length*sizeof(float), hipMemcpyDeviceToHost);

    //Generate synthetic data
    float upper_envelope_1, lower_envelope_1;
    float upper_envelope_2, lower_envelope_2;
    for (int ii=0;ii<num_iter;ii++)
    {
        //Copy first flow
        hipMemcpy(d_sx, flow_1.sx, flow_1.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_sy, flow_1.sy, flow_1.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_dx, flow_1.dx, flow_1.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_dy, flow_1.dy, flow_1.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_L, flow_1.L, flow_1.length*sizeof(float), hipMemcpyHostToDevice);
        //Copy second flow
        hipMemcpy(d_sx + flow_1.length, flow_2.sx, flow_2.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_sy + flow_1.length, flow_2.sy, flow_2.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_dx + flow_1.length, flow_2.dx, flow_2.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_dy + flow_1.length, flow_2.dy, flow_2.length*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_L + flow_1.length, flow_2.L, flow_2.length*sizeof(float), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        // Shuffle flow labels
        shuffle_label<<< ceil((float)length/64) , 64 >>>(d_sx, d_sy, d_dx, d_dy, d_L, length);
        hipDeviceSynchronize();
        // Recalate spatial distance matrix and cross k functions for synth
        // data
        calculate_spatial_distance_matrix <<< ceil((float)length/128) , 128, shared_mem_size >>> (d_sx,d_sy,d_dx,d_dy,d_L,dist_matrix_gpu,length,alpha,func_type);
        calculate_cross_k_function<<< ceil((float)length/128) , 128 >>>(dist_matrix_gpu,sig_vec,radius,A,length,flow_1.length);
        hipMemcpy(sig_vec_cpu, sig_vec, length*sizeof(float), hipMemcpyDeviceToHost);
        // Find upper and lower significance values
        if(ii>0)
        {
            upper_envelope_1 = max(upper_envelope_1,get_max(sig_vec_cpu,0,flow_1.length));
            lower_envelope_1 = min(lower_envelope_1,get_min(sig_vec_cpu,0,flow_1.length));
            upper_envelope_2 = max(upper_envelope_2,get_max(sig_vec_cpu,flow_1.length,length));
            lower_envelope_2 = min(lower_envelope_2,get_min(sig_vec_cpu,flow_1.length,length));
        }
        else
        {
            upper_envelope_1 = get_max(sig_vec_cpu,0,flow_1.length);
            lower_envelope_1 = get_min(sig_vec_cpu,0,flow_1.length);
            upper_envelope_2 = get_max(sig_vec_cpu,flow_1.length,length);
            lower_envelope_2 = get_min(sig_vec_cpu,flow_1.length,length);
        }
    }
    
    //Return flows with k-value above significance threshold
    std::vector<int> output;
    for(int i=0;i<flow_1.length;i++)
    {
        if(sig_vec_real[i]>upper_envelope_1)
        {
            output.push_back(i);
        }
    }
    output.push_back(-1);
    for(int i=flow_1.length;i<length;i++)
    {
        if(sig_vec_real[i]>upper_envelope_2)
        {
            output.push_back(i-flow_1.length);
        }
    }
    return output;
}
